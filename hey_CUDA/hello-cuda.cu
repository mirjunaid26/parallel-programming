
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void helloCUDA() {
    printf("Hello, World! from thread %d\n", threadIdx.x);
}

int main() {
    // Launch the kernel with 1 block containing 10 threads
    helloCUDA<<<1, 10>>>();

    // Wait for all threads to finish executing
    hipDeviceSynchronize();

    return 0;
}
